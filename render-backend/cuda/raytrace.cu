#include "hip/hip_runtime.h"
#include "raytrace.cuh"

namespace vrt {

#define SUBROUTINE __device__ __forceinline__

SUBROUTINE float3 random_in_unit_sphere3(hiprandState *state) {
    float3 p;
    do {
        p = 2.0f * make_float3(hiprand_uniform(state), hiprand_uniform(state), hiprand_uniform(state)) - make_float3(1.0f, 1.0f, 1.0f);
    } while (dot3(p, p) >= 1.0f);
    return p;
}

struct HitRecord {
    float3 point;
    float3 normal;
    float3 color;
    float t, u, v;
    Triangle* triangle;
    int material_id;
};

SUBROUTINE bool triangle_intersect(const Triangle& tri, const Ray& r, HitRecord& rec) {
    float3 e1 = tri.v[1].position - tri.v[0].position;
    float3 e2 = tri.v[2].position - tri.v[0].position;
    float3 p = cross(r.direction, e2);
    float det = dot3(e1, p);
    if (det > -1e-6f && det < 1e-6f) return false;
    float inv_det = 1.0f / det;
    float3 tvec = r.origin - tri.v[0].position;
    float u = dot3(tvec, p) * inv_det;
    if (u < 0.0f || u > 1.0f) return false;
    float3 q = cross(tvec, e1);
    float v = dot3(r.direction, q) * inv_det;
    if (v < 0.0f || u + v > 1.0f) return false;
    float t = dot3(e2, q) * inv_det;
    if (t < 0.0f || t > rec.t) return false;
    rec.t = t; rec.u = u; rec.v = v;
    rec.point = r.at(t);
    return true;
}

SUBROUTINE bool object_intersect(const RenderObject& object, const Ray& r, HitRecord& rec) {
    // use a stack to traverse the BVH
    int stack[MAX_HIT_STACK_SIZE];
    int top = 1;
    int hit_triangle = -1;
    stack[0] = object.n_bvh_nodes - 1;
    while (top > 0) {
        int node_idx = stack[--top];
        auto& node = object.bvh[node_idx];
        float t_near, t_far;
        if (!node.intersect(r, t_near, t_far)) continue;
        if (t_near > rec.t) continue; // early termination
        if (node.is_leaf) {
            if (node.left >= 0 && triangle_intersect(object.triangles[node.left], r, rec)) {
                hit_triangle = node.left;
            }
            if (node.right >= 0 && triangle_intersect(object.triangles[node.right], r, rec)) {
                hit_triangle = node.right;
            }
        } else {
            if (node.left >= 0 && top < MAX_HIT_STACK_SIZE) {
                stack[top++] = node.left;
            }
            if (node.right >= 0 && top < MAX_HIT_STACK_SIZE) {
                stack[top++] = node.right;
            }
        }
    }
    if (hit_triangle >= 0) {
        rec.triangle = &object.triangles[hit_triangle];
        rec.material_id = rec.triangle->material_id;
        return true;
    }
    return false;
}

SUBROUTINE void triangle_interpolate(HitRecord& rec, const Material* materials) { // get the normal and color
    float u = rec.u, v = rec.v;
    float w = 1.0f - u - v;
    float3 norm = w * rec.triangle->v[0].normal + u * rec.triangle->v[1].normal + v * rec.triangle->v[2].normal;
    rec.normal = normalize(norm);
    auto& material = materials[rec.material_id];
    if (material.cu_array == nullptr) {
        rec.color = material.albedo;
    } else {
        float texcoord_x = w * rec.triangle->v[0].texcoord.x + u * rec.triangle->v[1].texcoord.x + v * rec.triangle->v[2].texcoord.x;
        float texcoord_y = w * rec.triangle->v[0].texcoord.y + u * rec.triangle->v[1].texcoord.y + v * rec.triangle->v[2].texcoord.y;
        float4 texel = tex2D<float4>(material.tex_obj, texcoord_x, texcoord_y);
        rec.color = make_float3(texel.x, texel.y, texel.z);
    }
}

SUBROUTINE float3 reflect(const float3& v, const float3& n) {
    return v - 2.0f * dot3(v, n) * n;
}

SUBROUTINE bool refract(const float3& v, const float3& n, float ni_over_nt, float3& refracted) {
    float dt = dot3(v, n);
    float discriminant = 1.0f - ni_over_nt * ni_over_nt * (1.0f - dt * dt);
    if (discriminant > 0.0f) {
        refracted = ni_over_nt * (v - n * dt) - n * sqrt(discriminant);
        return true;
    }
    return false;
}

SUBROUTINE float schlick(float cosine, float ref_idx) {
    float r0 = (1.0f - ref_idx) / (1.0f + ref_idx);
    r0 = r0 * r0;
    return r0 + (1.0f - r0) * pow(1.0f - cosine, 5);
}

#define RAY(dir) Ray(rec.point + 1e-4f * dir, dir)

SUBROUTINE bool material_scatter(
    hiprandState* rand, const Material& material, const Ray& r, const HitRecord& rec, Ray& wo
) {
    switch (material.type) {
    case Material::LAMBERTIAN: {
        float3 direction;
        // turn the normal into same hemisphere as the ray direction
        if (dot3(r.direction, rec.normal) > 0.0f) {
            direction = -rec.normal + random_in_unit_sphere3(rand);
        } else {
            direction = rec.normal + random_in_unit_sphere3(rand);
        }
        wo = RAY(direction);
        return true;
    }
    case Material::METAL: {
        float3 direction = reflect(r.direction, rec.normal) 
            + material.metal_fuzz * random_in_unit_sphere3(rand);
        wo = RAY(direction);
        return dot3(wo.direction, rec.normal) > 0.0f;
    }
    case Material::REFLECTIVE: {
        float3 direction = reflect(r.direction, rec.normal);
        wo = RAY(direction);
        return true;
    }
    case Material::REFRACTIVE: {
        float3 outward_normal;
        float ni_over_nt;
        float cosine;
        float3 direction;
        if (dot3(r.direction, rec.normal) > 0.0f) {
            outward_normal = -rec.normal;
            ni_over_nt = material.optical_density;
            cosine = material.optical_density * dot3(r.direction, rec.normal);
        } else {
            outward_normal = rec.normal;
            ni_over_nt = 1.0f / material.optical_density;
            cosine = -dot3(r.direction, rec.normal);
        }
        float reflect_prob;
        if (refract(r.direction, outward_normal, ni_over_nt, direction)) {
            reflect_prob = schlick(cosine, material.optical_density);
        } else {
            reflect_prob = 1.0f;
        }
        if (hiprand_uniform(rand) < reflect_prob) {
            direction = reflect(r.direction, rec.normal);
        }
        wo = RAY(direction);
        return true;
    }
    }
    return false;
}

__global__ void raytrace_kernel(
    hiprandState* randstate, int n_randstate, int spp, float3 ambient,
    int max_depth, float russian_roulette,
    int n_rays, Ray* rays,
    int n_objects, const RenderObject* objects,
    int n_materials, const Material* materials,
    float3* output_buffer
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_rays) return;
    hiprandState* rand = &randstate[i % n_randstate];
    Ray ray = rays[i];
    float3 color = make_float3(1.0f, 1.0f, 1.0f);
    while (max_depth--) {
        HitRecord rec;
        rec.t = INFINITY;
        bool hit = false;
        for (int j = 0; j < n_objects; j++) {
            hit |= object_intersect(objects[j], ray, rec);
        }
        if (!hit) {
            color *= ambient;
            break;
        }
        triangle_interpolate(rec, materials);
        // debug: return abs normal as color
        // color = make_float3(abs(rec.normal.x), abs(rec.normal.y), abs(rec.normal.z));
        // break;
        auto& material = materials[rec.material_id];
        if (material.type == Material::LIGHT) {
            color *= rec.color;
            break;
        }
        else if (hiprand_uniform(rand) < russian_roulette) {
            color *= rec.color;
            Ray new_ray;
            if (!material_scatter(rand, material, ray, rec, new_ray)) {
                break;
            }
            color *= abs(dot3(new_ray.direction, rec.normal)); // Lambertian BRDF
            ray = new_ray;
        }
        else {
            color = make_float3(0.0f, 0.0f, 0.0f);
            break;
        }
    }
    // atmoically add the color to the output buffer
    int buffer_idx = i / spp;
    atomicAdd(&output_buffer[buffer_idx].x, color.x);
    atomicAdd(&output_buffer[buffer_idx].y, color.y);
    atomicAdd(&output_buffer[buffer_idx].z, color.z);
}

__global__ void post_process_kernel(int n_pixels, int spp, float alpha, float3* output_buffer) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_pixels) return;
    output_buffer[i] = output_buffer[i] * (alpha / spp);
    // clamp the color
    output_buffer[i].x = min(1.0f, max(0.0f, output_buffer[i].x));
    output_buffer[i].y = min(1.0f, max(0.0f, output_buffer[i].y));
    output_buffer[i].z = min(1.0f, max(0.0f, output_buffer[i].z));
}

__host__ void raytrace(
    hiprandState* randstate, int n_randstate, int spp,
    int max_depth, float alpha, float3 ambient,  float russian_roulette,
    int n_rays, Ray* rays,
    int n_objects, const RenderObject* objects,
    int n_materials, const Material* materials,
    float3* output_buffer
) {
    int block_size = 64, n_pixels = n_rays / spp;
    int num_blocks = (n_rays + block_size - 1) / block_size;
    raytrace_kernel<<<num_blocks, block_size>>>(
        randstate, n_randstate, spp, ambient, max_depth, russian_roulette,
        n_rays, rays, n_objects, objects, n_materials, materials,
        output_buffer
    );
    hipDeviceSynchronize();
    // print the output buffer to file
    #ifdef VRT_DEBUG
    float3* output = new float3[n_pixels];
    hipMemcpy(output, output_buffer, n_pixels * sizeof(float3), hipMemcpyDeviceToHost);
    for (int i = 0; i < n_pixels; i++) {
        printf("%d: (%f, %f, %f)\n", i, output[i].x, output[i].y, output[i].z);
    }
    delete[] output;
    #endif
    block_size = 256;
    num_blocks = (n_pixels + block_size - 1) / block_size;
    post_process_kernel<<<num_blocks, block_size>>>(
        n_pixels, spp, alpha, output_buffer
    );
    hipDeviceSynchronize();
}

}